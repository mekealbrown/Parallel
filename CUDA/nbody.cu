#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define CUDA_CHECK(call) { hipError_t err = call; if (err != hipSuccess) { fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); exit(1); } }

#define WINDOW_WIDTH 1280
#define WINDOW_HEIGHT 720
#define MAX_BODIES 8192
#define SOFTENING 0.1f
#define DAMPING 0.999f

// Structure for 3D vectors
typedef struct {
    float x, y, z;
} Vec3;

// Structure for body data
typedef struct {
    Vec3 position;
    Vec3 velocity;
    float mass;
} Body;

// Device arrays
Body *d_bodies;
float *d_positions; // For OpenGL VBO
struct hipGraphicsResource *cuda_vbo_resource;

// Simulation parameters
int num_bodies = 1000;
float dt = 0.01f;
float G = 1.0f;
int running = 0;
int editing = 0; // 0: not editing, 1: num_bodies, 2: dt, 3: G
char input_buffer[32] = "";
int input_pos = 0;

// OpenGL variables
GLuint vbo, vao;
GLFWwindow *window;

// CUDA kernel to compute forces and update positions
__global__ void compute_forces(Body *bodies, int n, float dt, float G, float softening, float damping) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    Vec3 force = {0.0f, 0.0f, 0.0f};
    Vec3 pos_i = bodies[i].position;

    for (int j = 0; j < n; j++) {
        if (i == j) continue;
        Vec3 pos_j = bodies[j].position;
        float dx = pos_j.x - pos_i.x;
        float dy = pos_j.y - pos_i.y;
        float dz = pos_j.z - pos_i.z;
        float dist = sqrtf(dx*dx + dy*dy + dz*dz + softening);
        float inv_dist3 = 1.0f / (dist * dist * dist);
        float force_mag = G * bodies[i].mass * bodies[j].mass * inv_dist3;
        force.x += force_mag * dx;
        force.y += force_mag * dy;
        force.z += force_mag * dz;
    }

    bodies[i].velocity.x = bodies[i].velocity.x * damping + (force.x / bodies[i].mass) * dt;
    bodies[i].velocity.y = bodies[i].velocity.y * damping + (force.y / bodies[i].mass) * dt;
    bodies[i].velocity.z = bodies[i].velocity.z * damping + (force.z / bodies[i].mass) * dt;

    bodies[i].position.x += bodies[i].velocity.x * dt;
    bodies[i].position.y += bodies[i].velocity.y * dt;
    bodies[i].position.z += bodies[i].velocity.z * dt;
}

// CUDA kernel to update VBO positions
__global__ void update_vbo(Body *bodies, float *positions, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    positions[3*i] = bodies[i].position.x;
    positions[3*i + 1] = bodies[i].position.y;
    positions[3*i + 2] = bodies[i].position.z;
}

// Initialize bodies with random positions and velocities
void init_bodies(Body *bodies, int n) {
    for (int i = 0; i < n; i++) {
        bodies[i].position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        bodies[i].position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        bodies[i].position.z = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        bodies[i].velocity.x = 0.1f * (rand() / (float)RAND_MAX - 0.5f);
        bodies[i].velocity.y = 0.1f * (rand() / (float)RAND_MAX - 0.5f);
        bodies[i].velocity.z = 0.1f * (rand() / (float)RAND_MAX - 0.5f);
        bodies[i].mass = 1.0f;
    }
}

// OpenGL setup
void init_opengl() {
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "Failed to initialize GLEW\n");
        exit(1);
    }

    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, num_bodies * 3 * sizeof(float), NULL, GL_DYNAMIC_DRAW);

    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);

    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard));

    glPointSize(2.0f);
    glEnable(GL_DEPTH_TEST);
}

// Shader setup
GLuint create_shader_program() {
    const char *vertex_shader_src =
        "#version 330 core\n"
        "layout(location = 0) in vec3 aPos;\n"
        "uniform mat4 mvp;\n"
        "void main() {\n"
        "    gl_Position = mvp * vec4(aPos, 1.0);\n"
        "}\n";

    const char *fragment_shader_src =
        "#version 330 core\n"
        "out vec4 FragColor;\n"
        "void main() {\n"
        "    FragColor = vec4(1.0, 1.0, 1.0, 1.0);\n"
        "}\n";

    GLuint vertex_shader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertex_shader, 1, &vertex_shader_src, NULL);
    glCompileShader(vertex_shader);

    GLuint fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragment_shader, 1, &fragment_shader_src, NULL);
    glCompileShader(fragment_shader);

    GLuint program = glCreateProgram();
    glAttachShader(program, vertex_shader);
    glAttachShader(program, fragment_shader);
    glLinkProgram(program);

    glDeleteShader(vertex_shader);
    glDeleteShader(fragment_shader);
    return program;
}

// Simple text rendering (minimal, using OpenGL quads)
void render_text(const char *text, float x, float y, float scale) {
    static const float char_width = 8.0f / WINDOW_WIDTH;
    static const float char_height = 16.0f / WINDOW_HEIGHT;
    glUseProgram(0);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, WINDOW_WIDTH, WINDOW_HEIGHT, 0, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glBegin(GL_QUADS);
    glColor3f(1.0f, 1.0f, 1.0f);
    for (int i = 0; text[i]; i++) {
        if (text[i] == '>') glColor3f(0.0f, 1.0f, 0.0f); // Highlight selected
        else glColor3f(1.0f, 1.0f, 1.0f);
        float cx = x + i * char_width * scale;
        float cy = y;
        glVertex2f(cx, cy);
        glVertex2f(cx + char_width * scale, cy);
        glVertex2f(cx + char_width * scale, cy + char_height * scale);
        glVertex2f(cx, cy + char_height * scale);
    }
    glEnd();
}

// Render menu
void render_menu() {
    char buf[128];
    snprintf(buf, sizeof(buf), "%sNum Bodies: %d", editing == 1 ? "> " : "  ", num_bodies);
    render_text(buf, 10.0f / WINDOW_WIDTH, 10.0f / WINDOW_HEIGHT, 1.0f);
    snprintf(buf, sizeof(buf), "%sTime Step: %.4f", editing == 2 ? "> " : "  ", dt);
    render_text(buf, 10.0f / WINDOW_WIDTH, 30.0f / WINDOW_HEIGHT, 1.0f);
    snprintf(buf, sizeof(buf), "%sGrav Constant: %.2f", editing == 3 ? "> " : "  ", G);
    render_text(buf, 10.0f / WINDOW_WIDTH, 50.0f / WINDOW_HEIGHT, 1.0f);
    snprintf(buf, sizeof(buf), "Input: %s", input_buffer);
    render_text(buf, 10.0f / WINDOW_WIDTH, 70.0f / WINDOW_HEIGHT, 1.0f);
    render_text("Press 1-3 to select, Enter to apply, Space to start", 10.0f / WINDOW_WIDTH, 90.0f / WINDOW_HEIGHT, 1.0f);
}

// Main rendering loop
void render(GLuint program, float time) {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    float cam_dist = 5.0f;
    float cam_x = cam_dist * sinf(time * 0.1f);
    float cam_z = cam_dist * cosf(time * 0.1f);
    float view[16] = {
        1.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 1.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 1.0f, 0.0f,
        -cam_x, 0.0f, -cam_z, 1.0f
    };
    float proj[16] = {
        1.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 1.0f, 0.0f, 0.0f,
        0.0f, 0.0f, -1.01f, -1.0f,
        0.0f, 0.0f, -0.2f, 0.0f
    };
    float mvp[16];
    for (int i = 0; i < 4; i++)
        for (int j = 0; j < 4; j++) {
            mvp[i*4 + j] = 0.0f;
            for (int k = 0; k < 4; k++)
                mvp[i*4 + j] += proj[i*4 + k] * view[k*4 + j];
        }

    glUseProgram(program);
    glUniformMatrix4fv(glGetUniformLocation(program, "mvp"), 1, GL_FALSE, mvp);

    glBindVertexArray(vao);
    glDrawArrays(GL_POINTS, 0, num_bodies);
    glBindVertexArray(0);

    render_menu();
}

// Key callback for parameter input
void key_callback(GLFWwindow *window, int key, int scancode, int action, int mods) {
    if (action != GLFW_PRESS) return;

    if (key == GLFW_KEY_1) editing = 1, input_buffer[0] = '\0', input_pos = 0;
    else if (key == GLFW_KEY_2) editing = 2, input_buffer[0] = '\0', input_pos = 0;
    else if (key == GLFW_KEY_3) editing = 3, input_buffer[0] = '\0', input_pos = 0;
    else if (key == GLFW_KEY_SPACE) {
        running = 1;
        editing = 0;
        input_buffer[0] = '\0';
        input_pos = 0;
    }
    else if (editing && key == GLFW_KEY_ENTER) {
        if (input_pos > 0) {
            if (editing == 1) {
                int new_num = atoi(input_buffer);
                if (new_num > 0 && new_num <= MAX_BODIES) num_bodies = new_num;
            }
            else if (editing == 2) {
                float new_dt = atof(input_buffer);
                if (new_dt > 0.0f) dt = new_dt;
            }
            else if (editing == 3) {
                float new_G = atof(input_buffer);
                if (new_G > 0.0f) G = new_G;
            }
        }
        input_buffer[0] = '\0';
        input_pos = 0;
        editing = 0;
    }
    else if (editing && key >= GLFW_KEY_0 && key <= GLFW_KEY_9) {
        if (input_pos < 31) input_buffer[input_pos++] = '0' + (key - GLFW_KEY_0), input_buffer[input_pos] = '\0';
    }
    else if (editing && key == GLFW_KEY_PERIOD) {
        if (input_pos < 31) input_buffer[input_pos++] = '.', input_buffer[input_pos] = '\0';
    }
    else if (editing && key == GLFW_KEY_BACKSPACE && input_pos > 0) {
        input_buffer[--input_pos] = '\0';
    }
}

// Main function
int main() {
    if (!glfwInit()) {
        fprintf(stderr, "Failed to initialize GLFW\n");
        return -1;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(WINDOW_WIDTH, WINDOW_HEIGHT, "3D N-Body Simulation", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);
    glfwSetKeyCallback(window, key_callback);

    init_opengl();
    GLuint shader_program = create_shader_program();

    CUDA_CHECK(hipMalloc(&d_bodies, MAX_BODIES * sizeof(Body)));

    Body *h_bodies = (Body*)malloc(MAX_BODIES * sizeof(Body));
    init_bodies(h_bodies, num_bodies);
    CUDA_CHECK(hipMemcpy(d_bodies, h_bodies, num_bodies * sizeof(Body), hipMemcpyHostToDevice));
    free(h_bodies);

    int threads_per_block = 256;
    int blocks = (num_bodies + threads_per_block - 1) / threads_per_block;

    float time = 0.0f;

    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        if (running) {
            compute_forces<<<blocks, threads_per_block>>>(d_bodies, num_bodies, dt, G, SOFTENING, DAMPING);
            CUDA_CHECK(hipDeviceSynchronize());

            float *d_vbo;
            size_t size;
            CUDA_CHECK(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
            CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&d_vbo, &size, cuda_vbo_resource));
            update_vbo<<<blocks, threads_per_block>>>(d_bodies, d_vbo, num_bodies);
            CUDA_CHECK(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

            if (running == 1) {
                running = 2;
                h_bodies = (Body*)malloc(MAX_BODIES * sizeof(Body));
                init_bodies(h_bodies, num_bodies);
                CUDA_CHECK(hipMemcpy(d_bodies, h_bodies, num_bodies * sizeof(Body), hipMemcpyHostToDevice));
                free(h_bodies);
                blocks = (num_bodies + threads_per_block - 1) / threads_per_block;
            }
        }

        render(shader_program, time);
        time += dt;

        glfwSwapBuffers(window);
    }

    CUDA_CHECK(hipFree(d_bodies));
    CUDA_CHECK(hipGraphicsUnregisterResource(cuda_vbo_resource));
    glDeleteBuffers(1, &vbo);
    glDeleteVertexArrays(1, &vao);
    glDeleteProgram(shader_program);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}